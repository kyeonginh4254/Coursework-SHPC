#include <mpi.h>
#include <cstdio>
#include <omp.h>
#include "layer.h"
#include "model.h"

#define NODE_SIZE 4096
#define BATCH_SIZE 1024
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// Timer macro for CUDA
#define START_TIMER(start) hipEventRecord(start, 0)
#define STOP_TIMER(start, stop, elapsed)                        \
  hipEventRecord(stop, 0);                                     \
  hipEventSynchronize(stop);                                   \
  hipEventElapsedTime(&elapsed, start, stop)

/* [Model Parameters]
 * _w: Weight parameter
 * _b: Bias parameter
 */
Parameter *emb_w;
Parameter *conv0_w, *conv0_b;
Parameter *conv1_w, *conv1_b;
Parameter *conv2_w, *conv2_b;
Parameter *conv3_w, *conv3_b;
Parameter *linear0_w, *linear0_b;
Parameter *linear1_w, *linear1_b;
Parameter *linear2_w, *linear2_b;
Parameter *linear3_w, *linear3_b;

void alloc_and_set_parameters(float *param, size_t param_size) {
  size_t pos = 0;

  emb_w = new Parameter({21635, 4096}, param + pos);
  pos += 21635 * 4096; 

  conv0_w = new Parameter({1024, 4096, 3}, param + pos);
  pos += 1024 * 4096 * 3; 
  conv0_b = new Parameter({1024}, param + pos);
  pos += 1024;

  conv1_w = new Parameter({1024, 4096, 5}, param + pos);
  pos += 1024 * 4096 * 5; 
  conv1_b = new Parameter({1024}, param + pos);
  pos += 1024;

  conv2_w = new Parameter({1024, 4096, 7}, param + pos);
  pos += 1024 * 4096 * 7;
  conv2_b = new Parameter({1024}, param + pos);
  pos += 1024;

  conv3_w = new Parameter({1024, 4096, 9}, param + pos);
  pos += 1024 * 4096 * 9;
  conv3_b = new Parameter({1024}, param + pos);
  pos += 1024;

  linear0_w = new Parameter({2048, 4096}, param + pos);
  pos += 2048 * 4096;
  linear0_b = new Parameter({2048}, param + pos);
  pos += 2048;

  linear1_w = new Parameter({1024, 2048}, param + pos);
  pos += 1024 * 2048;
  linear1_b = new Parameter({1024}, param + pos);
  pos += 1024;

  linear2_w = new Parameter({512, 1024}, param + pos);
  pos += 512 * 1024;
  linear2_b = new Parameter({512}, param + pos);
  pos += 512;

  linear3_w = new Parameter({2, 512}, param + pos);
  pos += 2 * 512;
  linear3_b = new Parameter({2}, param + pos);
  pos += 2;

  if (pos != param_size) {
    fprintf(stderr, "Parameter size mismatched: %zu != %zu\n", 
            pos, param_size);
    exit(EXIT_FAILURE);
  }
}

void free_parameters() {
  delete emb_w;
  delete conv0_w;
  delete conv0_b;
  delete conv1_w;
  delete conv1_b;
  delete conv2_w;
  delete conv2_b;
  delete conv3_w;
  delete conv3_b;
  delete linear0_w;
  delete linear0_b;
  delete linear1_w;
  delete linear1_b;
  delete linear2_w;
  delete linear2_b;
  delete linear3_w;
  delete linear3_b;
}

/* [Model Activations] 
 * _a: Activation buffer
 */
Activation *emb_a;
Activation *permute_a;
Activation *conv0_a, *relu0_a, *pool0_a;
Activation *conv1_a, *relu1_a, *pool1_a;
Activation *conv2_a, *relu2_a, *pool2_a;
Activation *conv3_a, *relu3_a, *pool3_a;
Activation *concat_a;
Activation *linear0_a, *linear1_a, *linear2_a, *linear3_a;

void alloc_activations() {
  emb_a = new Activation({BATCH_SIZE, SEQ_LEN, 4096});
  permute_a = new Activation({BATCH_SIZE, 4096, SEQ_LEN});
  conv0_a = new Activation({BATCH_SIZE, 1024, SEQ_LEN - 2});
  pool0_a = new Activation({BATCH_SIZE, 1024});
  conv1_a = new Activation({BATCH_SIZE, 1024, SEQ_LEN - 4});
  pool1_a = new Activation({BATCH_SIZE, 1024});
  conv2_a = new Activation({BATCH_SIZE, 1024, SEQ_LEN - 6});
  pool2_a = new Activation({BATCH_SIZE, 1024});
  conv3_a = new Activation({BATCH_SIZE, 1024, SEQ_LEN - 8});
  pool3_a = new Activation({BATCH_SIZE, 1024});
  concat_a = new Activation({BATCH_SIZE, 4096});
  linear0_a = new Activation({BATCH_SIZE, 2048});
  linear1_a = new Activation({BATCH_SIZE, 1024});
  linear2_a = new Activation({BATCH_SIZE, 512});
  linear3_a = new Activation({BATCH_SIZE, 2});
}

void free_activations() {
  delete emb_a;
  delete permute_a;
  delete conv0_a;
  delete pool0_a;
  delete conv1_a;
  delete pool1_a;
  delete conv2_a;
  delete pool2_a;
  delete conv3_a;
  delete pool3_a;
  delete concat_a;
  delete linear0_a;
  delete linear1_a;
  delete linear2_a;
  delete linear3_a;
}

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <string.h>

/* CONV MACRO */

#define conv_BATCH_SIZE 32

#define C 4096
#define OC 1024

#define K3 3
#define K5 5
#define K7 7
#define K9 9

#define BS 32

#define BOS3 BS - K3 + 1
#define BOS5 BS - K5 + 1
#define BOS7 BS - K7 + 1
#define BOS9 BS - K9 + 1

#define BOC 8
#define TC 8

/* LINEAR MACRO */

#define M0 4096
#define N0 2048

#define M1 2048
#define N1 1024

#define M2 1024
#define N2 512

#define M3 512
#define N3 2

#define NUM_DEVICES 4

// Function to predict sentiment
void predict_sentiment(int *inputs, float *outputs, size_t n_samples) {

  int mpi_rank, mpi_size;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);

  int *local_inputs = (int*)malloc(NODE_SIZE * SEQ_LEN * sizeof(int));
  float *local_outputs = (float*)malloc(NODE_SIZE * 2 * sizeof(float));

  MPI_Scatter(inputs, (int)(NODE_SIZE * SEQ_LEN), MPI_INT,
              local_inputs, (int)(NODE_SIZE * SEQ_LEN), MPI_INT,
              0, MPI_COMM_WORLD);

  size_t DEVICE_SAMPLES = NODE_SIZE / NUM_DEVICES;  

  Parameter* conv_w[4] = {conv0_w, conv1_w, conv2_w, conv3_w};
  Parameter* conv_b[4] = {conv0_b, conv1_b, conv2_b, conv3_b};
  size_t conv_K[4] = {3, 5, 7, 9};

  Parameter* linear_w[4] = {linear0_w, linear1_w, linear2_w, linear3_w};
  Parameter* linear_b[4] = {linear0_b, linear1_b, linear2_b, linear3_b};
  size_t linear_M[4] = {M0, M1, M2, M3};
  size_t linear_N[4] = {N0, N1, N2, N3};
  size_t os[4] = {14, 12, 10, 8};

  float *d_w[NUM_DEVICES];
  float *d_conv_w[NUM_DEVICES][4];
  float *d_conv_b[NUM_DEVICES][4];
  float *d_linear_w[NUM_DEVICES][4];
  float *d_linear_b[NUM_DEVICES][4];

  int *d_inputs[NUM_DEVICES];
  float *d_out_permuted[NUM_DEVICES];
  float *d_concat_a[NUM_DEVICES];
  float *d_out[NUM_DEVICES][4];
  float *d_conv_a[NUM_DEVICES][4];
  float *d_pool_a[NUM_DEVICES][4];
  float *d_linear_a[NUM_DEVICES][4];

  hipStream_t streams[NUM_DEVICES][4];
  omp_set_num_threads(NUM_DEVICES);

  #pragma omp parallel
  {
    int i = omp_get_thread_num();
    CHECK_CUDA(hipSetDevice(i));

    CHECK_CUDA(hipMalloc(&d_w[i], emb_w->shape[0] * emb_w->shape[1] * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_w[i], emb_w->buf, emb_w->shape[0] * emb_w->shape[1] * sizeof(float), hipMemcpyHostToDevice));

    for (size_t j = 0; j < 4; ++j) {
      CHECK_CUDA(hipMalloc(&d_conv_w[i][j], OC * C * conv_K[j] * sizeof(float)));
      CHECK_CUDA(hipMalloc(&d_conv_b[i][j], OC * sizeof(float)));
      CHECK_CUDA(hipMemcpy(d_conv_w[i][j], conv_w[j]->buf, OC * C * conv_K[j] * sizeof(float), hipMemcpyHostToDevice));
      CHECK_CUDA(hipMemcpy(d_conv_b[i][j], conv_b[j]->buf, OC * sizeof(float), hipMemcpyHostToDevice));
    }

    for (size_t j = 0; j < 4; ++j) {
      CHECK_CUDA(hipMalloc(&d_linear_w[i][j], linear_M[j] * linear_N[j] * sizeof(float)));
      CHECK_CUDA(hipMalloc(&d_linear_b[i][j], linear_M[j] * sizeof(float)));
      CHECK_CUDA(hipMemcpy(d_linear_w[i][j], linear_w[j]->buf, linear_M[j] * linear_N[j] * sizeof(float), hipMemcpyHostToDevice));
      CHECK_CUDA(hipMemcpy(d_linear_b[i][j], linear_b[j]->buf, linear_M[j] * sizeof(float), hipMemcpyHostToDevice));
    }

    CHECK_CUDA(hipMalloc(&d_inputs[i], BATCH_SIZE * SEQ_LEN * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_out_permuted[i], BATCH_SIZE * 4096 * SEQ_LEN * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_concat_a[i], BATCH_SIZE * 4096 * sizeof(float)));

    for (size_t j = 0; j < 4; ++j) {
      CHECK_CUDA(hipMalloc(&d_out[i][j], BATCH_SIZE * OC * os[j] * sizeof(float)));
      CHECK_CUDA(hipMalloc(&d_conv_a[i][j], BATCH_SIZE * OC * os[j] * sizeof(float)));
      CHECK_CUDA(hipMalloc(&d_pool_a[i][j], BATCH_SIZE * OC * sizeof(float)));
      CHECK_CUDA(hipMalloc(&d_linear_a[i][j], BATCH_SIZE * linear_N[j] * sizeof(float)));
    }

    for (size_t j = 0; j < 4; j++) {
      CHECK_CUDA(hipStreamCreate(&streams[i][j]));
    }
  }

  #pragma omp parallel for
  for (size_t i = 0; i < NUM_DEVICES; ++i) {
    CHECK_CUDA(hipSetDevice(i));

    size_t DEVICE_OFFSET = DEVICE_SAMPLES * i;

    for (size_t GPU_OFFSET = 0; GPU_OFFSET < DEVICE_SAMPLES; GPU_OFFSET += BATCH_SIZE) {

      int* batch_input_ptr = local_inputs + (DEVICE_OFFSET + GPU_OFFSET) * SEQ_LEN;
      CHECK_CUDA(hipMemcpyAsync(d_inputs[i], batch_input_ptr, 
                                  BATCH_SIZE * SEQ_LEN * sizeof(int), 
                                  hipMemcpyHostToDevice, streams[i][0]));

      dim3 block(32, 32);
      dim3 grid(CEIL_DIV(4096, 32), CEIL_DIV(SEQ_LEN, 32), BATCH_SIZE);
      Embedding_Permute_Kernel<<<grid, block, 0, streams[i][0]>>>(d_inputs[i], d_w[i], d_out_permuted[i], BATCH_SIZE, SEQ_LEN, 4096);

      Conv1d(d_out_permuted[i], d_conv_w[i], d_conv_b[i], d_conv_a[i], d_out[i], streams[i]);

      for (size_t j = 0; j < 4; j++) {
        GetMax(d_conv_a[i][j], d_pool_a[i][j], os[j], streams[i][j]);
      }

      Concat(d_pool_a[i][0], d_pool_a[i][1], d_pool_a[i][2], d_pool_a[i][3], d_concat_a[i], streams[i][0]);
      Linear_ReLU_CUDA(d_concat_a[i], d_linear_w[i][0], d_linear_b[i][0], d_linear_a[i][0], 4096, 2048, streams[i][0]);
      Linear_ReLU_CUDA(d_linear_a[i][0], d_linear_w[i][1], d_linear_b[i][1], d_linear_a[i][1], 2048, 1024, streams[i][0]);
      Linear_ReLU_CUDA(d_linear_a[i][1], d_linear_w[i][2], d_linear_b[i][2], d_linear_a[i][2], 1024, 512, streams[i][0]);
      Linear_CUDA(d_linear_a[i][2], d_linear_w[i][3], d_linear_b[i][3], d_linear_a[i][3], 512, 2, streams[i][0]);

      CHECK_CUDA(hipMemcpyAsync(local_outputs + (DEVICE_OFFSET + GPU_OFFSET) * 2,
                                  d_linear_a[i][3], BATCH_SIZE * 2 * sizeof(float),
                                  hipMemcpyDeviceToHost, streams[i][0]));
    }
  } // end of omp parallel

  // 모든 GPU 연산 완료 대기
  for (size_t i = 0; i < NUM_DEVICES; ++i) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamSynchronize(streams[i][0]));
  }

  MPI_Gather(local_outputs, (int)(NODE_SIZE * 2), MPI_FLOAT,
             outputs, (int)(NODE_SIZE * 2), MPI_FLOAT,
             0, MPI_COMM_WORLD);

  // Free Host memory
  free(local_inputs);
  free(local_outputs);

  // Free CUDA resources
  #pragma omp parallel for
  for (size_t i = 0; i < NUM_DEVICES; ++i) {
    CHECK_CUDA(hipSetDevice(i));
    
    // Free embedding weight
    CHECK_CUDA(hipFree(d_w[i]));
    
    // Free conv weights and biases
    for (size_t j = 0; j < 4; ++j) {
      CHECK_CUDA(hipFree(d_conv_w[i][j]));
      CHECK_CUDA(hipFree(d_conv_b[i][j]));
    }

    // Free linear weights and biases
    for (size_t j = 0; j < 4; ++j) {
      CHECK_CUDA(hipFree(d_linear_w[i][j]));
      CHECK_CUDA(hipFree(d_linear_b[i][j]));
    }

    // Free inputs and intermediate buffers
    CHECK_CUDA(hipFree(d_inputs[i]));
    CHECK_CUDA(hipFree(d_out_permuted[i]));
    CHECK_CUDA(hipFree(d_concat_a[i]));

    // Free intermediate outputs of conv, pool, linear
    for (size_t j = 0; j < 4; ++j) {
      CHECK_CUDA(hipFree(d_out[i][j]));
      CHECK_CUDA(hipFree(d_conv_a[i][j]));
      CHECK_CUDA(hipFree(d_pool_a[i][j]));
      CHECK_CUDA(hipFree(d_linear_a[i][j]));
    }

    // Destroy streams
    for (size_t j = 0; j < 4; ++j) {
      CHECK_CUDA(hipStreamDestroy(streams[i][j]));
    }
  }
}
