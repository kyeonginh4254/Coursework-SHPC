#include "hip/hip_runtime.h"
#include "matmul_single.h"
#include "util.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#define TS 32
#define WPT 2
#define RTS TS/WPT

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                              int K) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  if (i >= M || j >= N)
    return;

  C[i * N + j] = 0;
  for (int k = 0; k < K; ++k) {
    C[i * N + j] += A[i * K + k] * B[k * N + j];
  }
}

// Array of device (GPU) pointers
static float *a_d;
static float *b_d;
static float *c_d;

void matmul(const float *A, const float *B, float *C, int M, int N, int K) {

  // Upload A and B matrix to every GPU
  CUDA_CALL(hipMemcpy(a_d, A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(b_d, B, K * N * sizeof(float), hipMemcpyHostToDevice));

  // Launch kernel on every GPU
  dim3 blockDim(1, 1, 1);
  dim3 gridDim(M, N, 1);

  matmul_kernel<<<gridDim, blockDim>>>(a_d, b_d, c_d, M, N, K);

  CUDA_CALL(hipDeviceSynchronize());

  // Download C matrix from GPUs
  CUDA_CALL(hipMemcpy(C, c_d, M * N * sizeof(float), hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CUDA_CALL(hipDeviceSynchronize());
}

void matmul_initialize(int M, int N, int K) {
  
  int num_devices;
  // Only root process do something
  CUDA_CALL(hipGetDeviceCount(&num_devices));

  if (num_devices <= 0) {
    printf("No CUDA device found. Aborting\n");
    exit(1);
  }

  // Allocate device memory 
  CUDA_CALL(hipMalloc(&a_d, M * K * sizeof(float)));
  CUDA_CALL(hipMalloc(&b_d, K * N * sizeof(float)));
  CUDA_CALL(hipMalloc(&c_d, M * N * sizeof(float)));
}

void matmul_finalize() {

  // Free GPU memory
  CUDA_CALL(hipFree(a_d));
  CUDA_CALL(hipFree(b_d));
  CUDA_CALL(hipFree(c_d));
}
