#include "hip/hip_runtime.h"
#include "matmul_multi.h"
#include "util.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#define TS 64
#define WPT 2
#define RTS TS/WPT

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }

__global__ void matmul_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    const int row = threadIdx.y; // Local row index within the block
    const int col = threadIdx.x; // Local column index within the block
    const int globalRow = RTS * blockIdx.y + row; // Global row index
    const int globalCol = RTS * blockIdx.x + col; // Global column index

    __shared__ float Asub[TS][TS];
    __shared__ float Bsub[TS][TS];


  float Cvalue[WPT][WPT] = {0.0f};

    const int numTiles = (K + TS - 1) / TS;

    for (int t = 0; t < numTiles; t++) {
        int tiledRow = RTS * t + row;
        int tiledCol = RTS * t + col;

        // Load data into shared memory
        for (int w1 = 0; w1 < WPT; ++w1) {
            for (int w2 = 0; w2 < WPT; ++w2) {
                int rowA = globalRow * WPT + w1;
                int colA = tiledCol * WPT + w2;
                if (rowA < M && colA < K) {
                    Asub[row * WPT + w1][col * WPT + w2] = A[rowA * K + colA];
                } else {
                    Asub[row * WPT + w1][col * WPT + w2] = 0.0f;
                }

                int rowB = tiledRow * WPT + w1;
                int colB = globalCol * WPT + w2;
                if (rowB < K && colB < N) {
                    Bsub[row * WPT + w1][col * WPT + w2] = B[rowB * N + colB];
                } else {
                    Bsub[row * WPT + w1][col * WPT + w2] = 0.0f;
                }
            }
        }

        __syncthreads();

        // Compute partial results
        for (int k = 0; k < TS; ++k) {
            for (int w1 = 0; w1 < WPT; ++w1) {
                for (int w2 = 0; w2 < WPT; w2++) {
                    Cvalue[w1][w2] += Asub[row * WPT + w1][k] * Bsub[k][col * WPT + w2];
                }
            }
        }

        __syncthreads();
    }

    // Write the final result to global memory
    for (int w1 = 0; w1 < WPT; ++w1) {
        for (int w2 = 0; w2 < WPT; ++w2) {
            int rowC = globalRow * WPT + w1;
            int colC = globalCol * WPT + w2;
            if (rowC < M && colC < N) {
                C[rowC * N + colC] = Cvalue[w1][w2];
            }
        }
    }
}

//////////////////////////

// global datas

#define MAX_NUM_GPU 4
int num_devices = 0;
#define CHK 1024
#define NCHK 4096/CHK

hipStream_t streams[MAX_NUM_GPU][NCHK];

static float *a_d[MAX_NUM_GPU][NCHK];
static float *b_d[MAX_NUM_GPU];
static float *c_d[MAX_NUM_GPU][NCHK];
static int Mbegin[MAX_NUM_GPU][NCHK], Mend[MAX_NUM_GPU][NCHK];
float *pinned_A = nullptr, *pinned_B = nullptr, *pinned_C = nullptr;

void matmul(const float *A, const float *B, float *C, int M, int N, int K) {

  int chk = CHK;
  int nchk = NCHK;

  memcpy(pinned_A, A, M * K * sizeof(float));
  memcpy(pinned_B, B, K * N * sizeof(float));

  for (int i = 0; i < num_devices; ++i) {
    CUDA_CALL(hipSetDevice(i));
    CUDA_CALL(hipMemcpyAsync(b_d[i],
                              pinned_B,
                              K * N * sizeof(float),
                              hipMemcpyHostToDevice,
                              streams[i][0]));

    CUDA_CALL(hipStreamSynchronize(streams[i][0]));

    for (int n = 0; n < nchk; n++) {

      CUDA_CALL(hipMemcpyAsync(a_d[i][n],
                                pinned_A + Mbegin[i][n] * K,
                                chk * K * sizeof(float),
                                hipMemcpyHostToDevice,
                                streams[i][n]));

      dim3 blockDim(RTS, RTS);
      dim3 gridDim((N + TS - 1) / TS, (chk + TS - 1) / TS);

      matmul_kernel<<<gridDim, blockDim, 0, streams[i][n]>>>(a_d[i][n], b_d[i], c_d[i][n], chk, N, K);

      CUDA_CALL(hipMemcpyAsync(pinned_C + Mbegin[i][n] * N,
                                c_d[i][n],
                                chk * N * sizeof(float),
                                hipMemcpyDeviceToHost,
                                streams[i][n]));

    }
  }

  memcpy(C, pinned_C, M * N * sizeof(float));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipDeviceSynchronize());
  }
}

void matmul_initialize(int M, int N, int K) {

  CUDA_CALL(hipHostAlloc((void**) &pinned_A, M * K * sizeof(float), hipHostMallocWriteCombined));
  CUDA_CALL(hipHostAlloc((void**) &pinned_B, K * N * sizeof(float), hipHostMallocWriteCombined));
  CUDA_CALL(hipHostAlloc((void**) &pinned_C, M * N * sizeof(float), hipHostMallocDefault));

  CUDA_CALL(hipGetDeviceCount(&num_devices));

  printf("Using %d devices\n", num_devices);
  for (int i = 0; i < num_devices; i++) {
    hipDeviceProp_t prop;
    CUDA_CALL(hipGetDeviceProperties(&prop, i));

    // Try printing more detailed information here
    printf("GPU %d: %s\n", i, prop.name);
  }

  if (num_devices <= 0) {
    printf("No CUDA device found. Aborting\n");
    exit(1);
  }

  // Setup problem size for each GPU
  for (int i = 0; i < num_devices; i++) {
    for (int n = 0; n < NCHK; n++) {
      Mbegin[i][n] = (M / num_devices) * i + CHK * n;
      Mend[i][n] = (M / num_devices) * i + CHK * (n + 1);
    }
  }

  // Allocate device memory for each GPU
  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipSetDevice(i));
    CUDA_CALL(hipMalloc(&b_d[i], K * N * sizeof(float)));
    for (int n = 0; n < NCHK; n++) {
      CUDA_CALL(hipStreamCreate(&streams[i][n]));
      CUDA_CALL(hipMalloc(&a_d[i][n], (Mend[i][n] - Mbegin[i][n]) * K * sizeof(float)));
      CUDA_CALL(hipMalloc(&c_d[i][n], (Mend[i][n] - Mbegin[i][n]) * N * sizeof(float)));
    }
  }
}

void matmul_finalize() {

  for (int i = 0; i < num_devices; i++) {
    CUDA_CALL(hipFree(b_d[i]));
    for (int n = 0; n < NCHK; n++) {
      CUDA_CALL(hipStreamDestroy(streams[i][n]));
      CUDA_CALL(hipFree(a_d[i][n]));
      CUDA_CALL(hipFree(c_d[i][n]));
    }
  }

  CUDA_CALL(hipHostFree(pinned_A));
  CUDA_CALL(hipHostFree(pinned_B));
  CUDA_CALL(hipHostFree(pinned_C)); 

}